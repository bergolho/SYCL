#include <iostream>
#include <hip/hip_runtime.h>
#include "vector_library.h"

const int N = 16;

int main(){

    float A[N] = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};
    float B[N] = {2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2, 2};
    float C[N] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    //# Allocate memory on device
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N*sizeof(float));
    hipMalloc(&d_B, N*sizeof(float));
    hipMalloc(&d_C, N*sizeof(float));

    //# copy vector data from host to device
    hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);

    //# sumbit task to compute VectorAdd on device
    VectorAddKernel<<<1, N>>>(d_A, d_B, d_C);

    //# copy result of vector data from device to host
    hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);

    //# print result on host
    for (int i = 0; i < N; i++) std::cout<< C[i] << " ";
    std::cout << "\n";

    //# free allocation on device
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}